
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <iterator>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 10
#define CHARS_PER_PASSWORD 30
#define M 11000
#define THREADS_PER_BLOCK 512
using namespace std;

__device__ char* findPassword(char *grid, int x, int n);
__device__ char* generatePassword(char *grid, char* domain, int domainSize, int n);
__device__ int randomNumber(int blockId);
__device__ int findCharIndex(char *grid, char toFind, int fromX, int fromY, int dir, int n);
__device__ int index(int x, int y, int n);

__global__ void findPasswords( char *grid, char *result, int n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index <= M) {
		char* arr = findPassword(grid, index, n);
		for (int i = 0; i < CHARS_PER_PASSWORD; i++) {
			result[(index * CHARS_PER_PASSWORD) + i] = arr[i];
		}
	}
}

__device__ char* findPassword(char *grid, int x, int n) {
	char * thisArr = new char[CHARS_PER_PASSWORD]();
	char domain[5];
	
	int domainSize = (sizeof(domain)/sizeof(char));
	for (int i = 0; i < domainSize; i++) {
		int rand = randomNumber(x) % n;
        	if (rand < 0) {
			// kinda hacky
			rand = rand * -1;
		}
		domain[i] = 'A' + rand;	
	}

	char * generatedPassword = generatePassword(grid, domain, domainSize, n);

	for (int i = 0; i < domainSize; i++) {
		thisArr[i] = domain[i];
	}

	thisArr[domainSize] = '-';
	thisArr[domainSize + 1] = '>';

	for (int i = 0; i < domainSize * 2; i++) {
		thisArr[domainSize + 2 + i] = generatedPassword[i];
	}

	return thisArr;
}

__device__ char* generatePassword(char *grid, char* domain, int domainSize, int n) {
	char* generatedPassword = new char[domainSize*2]();

	// x and y should probably be randomly initialized but they're not :)
	int x = 0;
	int y = 0;
	int dir = 0;

	for (int i = 0; i < domainSize; i++) {
		int charIndext = findCharIndex(grid, domain[i], x, y, dir, n);
		if (dir % 2 == 0) {
			x = charIndext;
		} else {
			y = charIndext;
		}
		dir++;
	}

	for (int i = 0; i < domainSize; i++) {
			int charIndex = findCharIndex(grid, domain[i], x, y, dir, n);
			int nextOne;
			int nextTwo;
			int nextThree;

			if ( dir % 2 == 0) {
				if (charIndex < x) {
					nextOne = charIndex - 1;
					nextTwo = charIndex - 2;
					nextThree = charIndex - 3;	
				} else {
					nextOne = charIndex + 1;
					nextTwo = charIndex + 2;
					nextThree = charIndex + 3;
				}
			} else {
				if (charIndex < y) {
					nextOne = charIndex - 1;
                                        nextTwo = charIndex - 2;
					nextThree = charIndex - 3;
				} else {
					nextOne = charIndex + 1;
                                        nextTwo = charIndex + 2;
					nextThree = charIndex + 3;
				}
			}

			if (nextOne >= n) {
				nextOne = nextOne - n;
			} else if (nextOne < 0) {
				nextOne = n + nextOne;
			}
			
			if (nextTwo >= n) {
                                nextTwo = nextTwo - n;
                        } else if (nextTwo < 0) {
                                nextTwo = n + nextTwo;
                        }

			if (nextThree >= n) {
                                nextThree = nextThree - n;
                        } else if (nextThree < 0) {
                                nextThree = n + nextThree;
                        }


			if (dir % 2 == 0) {
				if (i < domainSize - 1 && grid[index(nextTwo,y,n)] == domain[i + 1]) {
					x = nextThree;
				} else {
					x = nextTwo;
				}
				generatedPassword[i*2] = grid[index(nextOne, y, n)];
                		generatedPassword[(i*2) + 1] = grid[index(nextTwo,y,n)];
			} else {
				if (i < domainSize - 1 && grid[index(x,nextTwo,n)] == domain[i + 1]) {
					y = nextThree;
				} else {
					y = nextTwo;
				}
				generatedPassword[i*2] = grid[index(x,nextOne,n)];
                		generatedPassword[(i*2) + 1] = grid[index(x,nextTwo,n)];
			} 
			
		dir++;
	}
	
	return generatedPassword;
}

__device__ int findCharIndex(char *grid, char toFind, int fromX, int fromY, int dir, int n) {
	if (dir % 2 == 0) {
		for (int i = 0; i < n; i++) {
			if (grid[index(i,fromY,n)] == toFind) {
				return i;
			}
		}
	} else {
		for (int i = 0; i < n; i++) {
			if (grid[index(fromX, i, n)] == toFind) {
				return	i;
			}
		}
	}
	return 0;
}

__device__ int randomNumber(int blockId) {
	hiprandState_t state;
	hiprand_init(clock64(), blockId, 0, &state);
	return hiprand(&state);
}

__device__ int index(int x, int y, int n) {
	return n*y + x;
}

int main( void ) 
{
	int n;
	cout << "What is N?\n";
	cin >> n;
	cout << "n set to " + n; 
	
	ifstream mygridfile;
	mygridfile.open("grid.txt");
	
	char* grid;
	hipMallocManaged( (void**)&grid, n * n * sizeof(char));
	for (int i = 0; i < (n*n); i++) {
		mygridfile >> grid[i];
	}

	mygridfile.close();

	cout << "print here";
	for (int i = 0; i < n*n; i++) {
		cout << grid[i];
	}

	char* result;
	hipMallocManaged( (void**)&result, CHARS_PER_PASSWORD*sizeof(char)*M);
	findPasswords<<<M/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(grid, result, n);
	hipDeviceSynchronize();

	ofstream resultfile;
	resultfile.open("passwords.txt");

	// This isn't deduped but there's an extra 1,000 passwords for you!
	for( int i = 0 ; i < M*CHARS_PER_PASSWORD; i ++ ){
		if (result[i]!='\0') { 
			resultfile << result[i];
		}
		if ((i + 1) % 30 == 0) {
			resultfile << endl;
		} 
	}
	resultfile.close();

	return 0;
}
